#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"

//#include "pcl/gpu/utils/device/vector_math.hpp"

namespace pcl {
namespace device {
__global__ void initColorVolumeKernel(PtrStep<uchar4> volume) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < VOLUME_X && y < VOLUME_Y) {
        uchar4 *pos = volume.ptr(y) + x;
        int z_step = VOLUME_Y * volume.step / sizeof(*pos);

#pragma unroll
        for (int z = 0; z < VOLUME_Z; ++z, pos += z_step)
            *pos = make_uchar4(0, 0, 0, 0);
    }
}

__global__ void clearColorSliceKernel(PtrStep<uchar4> color_volume,
                                      pcl::gpu::tsdf_buffer buffer,
                                      int3 minBounds, int3 maxBounds) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // compute relative indices
    int idX, idY;

    if (x < minBounds.x)
        idX = x + buffer.voxels_size.x;
    else
        idX = x;

    if (y < minBounds.y)
        idY = y + buffer.voxels_size.y;
    else
        idY = y;

    if (x < buffer.voxels_size.x && y < buffer.voxels_size.y) {
        if ((idX >= minBounds.x && idX <= maxBounds.x) ||
            (idY >= minBounds.y && idY <= maxBounds.y)) {
            // BLACK ZONE => clear on all Z values

            /// Pointer to the first x,y,0
            uchar4 *pos = color_volume.ptr(y) + x;

            /// Get the step on Z
            int z_step =
                buffer.voxels_size.y * color_volume.step / sizeof(*pos);

            /// Get the size of the whole TSDF memory
            int size = buffer.color_memory_end - buffer.color_memory_start + 1;

            /// Move along z axis
#pragma unroll
            for (int z = 0; z < buffer.voxels_size.z; ++z, pos += z_step) {
                /// If we went outside of the memory, make sure we go back to
                /// the begining of it
                if (pos > buffer.color_memory_end)
                    pos = pos - size;

                *pos = make_uchar4(0, 0, 0, 0);
            }
        } else /* if( idX > maxBounds.x && idY > maxBounds.y)*/
        {

            /// RED ZONE  => clear only appropriate Z

            /// Pointer to the first x,y,0
            uchar4 *pos = color_volume.ptr(y) + x;

            /// Get the step on Z
            int z_step =
                buffer.voxels_size.y * color_volume.step / sizeof(*pos);

            /// Get the size of the whole TSDF memory
            int size = buffer.color_memory_end - buffer.color_memory_start + 1;

            /// Move pointer to the Z origin
            pos += minBounds.z * z_step;

            /// If the Z offset is negative, we move the pointer back
            if (maxBounds.z < 0)
                pos += maxBounds.z * z_step;

            /// We make sure that we are not already before the start of the
            /// memory
            if (pos < buffer.color_memory_start)
                pos = pos + size;

            int nbSteps = abs(maxBounds.z);

#pragma unroll
            for (int z = 0; z < nbSteps; ++z, pos += z_step) {
                /// If we went outside of the memory, make sure we go back to
                /// the begining of it
                if (pos > buffer.color_memory_end)
                    pos = pos - size;

                *pos = make_uchar4(0, 0, 0, 0);
            }
        } // else /* if( idX > maxBounds.x && idY > maxBounds.y)*/
    }     // if ( x < VOLUME_X && y < VOLUME_Y)
} // clearColorSliceKernel
} // namespace device
} // namespace pcl

void pcl::device::initColorVolume(PtrStep<uchar4> color_volume) {
    dim3 block(32, 16);
    dim3 grid(1, 1, 1);
    grid.x = divUp(VOLUME_X, block.x);
    grid.y = divUp(VOLUME_Y, block.y);

    initColorVolumeKernel<<<grid, block>>>(color_volume);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
}

namespace pcl {
namespace device {
struct ColorVolumeImpl {
    enum {
        CTA_SIZE_X = 32,
        CTA_SIZE_Y = 8,

        ONE_VOXEL = 0
    };

    Intr intr;

    PtrStep<float> vmap;
    PtrStepSz<uchar3> colors;

    Mat33 R_inv;
    float3 t;

    float3 cell_size;
    float tranc_dist;

    int max_weight;

    mutable PtrStep<uchar4> color_volume;

    __device__ __forceinline__ int3 getVoxel(float3 point) const {
        int vx =
            __float2int_rd(point.x / cell_size.x); // round to negative infinity
        int vy = __float2int_rd(point.y / cell_size.y);
        int vz = __float2int_rd(point.z / cell_size.z);

        return make_int3(vx, vy, vz);
    }

    __device__ __forceinline__ float3 getVoxelGCoo(int x, int y, int z) const {
        float3 coo = make_float3(x, y, z);
        coo += 0.5f; // shift to cell center;

        coo.x *= cell_size.x;
        coo.y *= cell_size.y;
        coo.z *= cell_size.z;

        return coo;
    }

    __device__ __forceinline__ void
    operator()(pcl::gpu::tsdf_buffer buffer) const {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= VOLUME_X || y >= VOLUME_Y)
            return;

        for (int z = 0; z < VOLUME_X; ++z) {
            float3 v_g = getVoxelGCoo(x, y, z);

            float3 v = R_inv * (v_g - t);

            if (v.z <= 0)
                continue;

            int2 coo; // project to current cam
            coo.x = __float2int_rn(v.x * intr.fx / v.z + intr.cx);
            coo.y = __float2int_rn(v.y * intr.fy / v.z + intr.cy);

            if (coo.x >= 0 && coo.y >= 0 && coo.x < colors.cols &&
                coo.y < colors.rows) {
                float3 p;
                p.x = vmap.ptr(coo.y)[coo.x];

                if (isnan(p.x))
                    continue;

                p.y = vmap.ptr(coo.y + colors.rows)[coo.x];
                p.z = vmap.ptr(coo.y + colors.rows * 2)[coo.x];

                bool update = false;
                if (ONE_VOXEL) {
                    int3 vp = getVoxel(p);
                    update = vp.x == x && vp.y == y && vp.z == z;
                } else {
                    float dist = norm(p - v_g);
                    update = dist < tranc_dist;
                }

                if (update) {
                    uchar4 *ptr = color_volume.ptr(VOLUME_Y * z + y) + x;

                    shift_color_pointer(&ptr, buffer);

                    uchar3 rgb = colors.ptr(coo.y)[coo.x];
                    uchar4 volume_rgbw = *ptr;

                    int weight_prev = volume_rgbw.w;

                    const float Wrk = 1.f;
                    float new_x = (volume_rgbw.x * weight_prev + Wrk * rgb.x) /
                                  (weight_prev + Wrk);
                    float new_y = (volume_rgbw.y * weight_prev + Wrk * rgb.y) /
                                  (weight_prev + Wrk);
                    float new_z = (volume_rgbw.z * weight_prev + Wrk * rgb.z) /
                                  (weight_prev + Wrk);

                    int weight_new = weight_prev + 1;

                    uchar4 volume_rgbw_new;
                    volume_rgbw_new.x = min(255, max(0, __float2int_rn(new_x)));
                    volume_rgbw_new.y = min(255, max(0, __float2int_rn(new_y)));
                    volume_rgbw_new.z = min(255, max(0, __float2int_rn(new_z)));
                    volume_rgbw_new.w = min(max_weight, weight_new);

                    *ptr = volume_rgbw_new;
                }
            } /* in camera image range */
        }     /* for(int z = 0; z < VOLUME_X; ++z) */
    }         /* void operator() */
};

__global__ void updateColorVolumeKernel(const ColorVolumeImpl cvi,
                                        pcl::gpu::tsdf_buffer buffer) {
    cvi(buffer);
}
} // namespace device
} // namespace pcl

void pcl::device::updateColorVolume(
    const Intr &intr, float tranc_dist, const Mat33 &R_inv, const float3 &t,
    const MapArr &vmap, const PtrStepSz<uchar3> &colors,
    const float3 &volume_size, PtrStep<uchar4> color_volume,
    pcl::gpu::tsdf_buffer *buffer, int max_weight) {
    ColorVolumeImpl cvi;
    cvi.vmap = vmap;
    cvi.colors = colors;
    cvi.color_volume = color_volume;

    cvi.R_inv = R_inv;
    cvi.t = t;
    cvi.intr = intr;
    cvi.tranc_dist = tranc_dist;
    cvi.max_weight = min(max(0, max_weight), 255);

    cvi.cell_size.x = volume_size.x / VOLUME_X;
    cvi.cell_size.y = volume_size.y / VOLUME_Y;
    cvi.cell_size.z = volume_size.z / VOLUME_Z;

    dim3 block(ColorVolumeImpl::CTA_SIZE_X, ColorVolumeImpl::CTA_SIZE_Y);
    dim3 grid(divUp(VOLUME_X, block.x), divUp(VOLUME_Y, block.y));

    updateColorVolumeKernel<<<grid, block>>>(cvi, *buffer);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
}

namespace pcl {
namespace device {
__global__ void extractColorsKernel(const float3 cell_size,
                                    const PtrStep<uchar4> color_volume,
                                    pcl::gpu::tsdf_buffer buffer,
                                    const PtrSz<PointType> points,
                                    uchar4 *colors) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < points.size) {
        int3 v;
        float3 p = *(const float3 *)(points.data + idx);
        v.x = __float2int_rd(p.x / cell_size.x); // round to negative infinity
        v.y = __float2int_rd(p.y / cell_size.y);
        v.z = __float2int_rd(p.z / cell_size.z);

        const uchar4 *tmp_pos =
            &(color_volume.ptr(buffer.voxels_size.y * v.z + v.y)[v.x]);
        uchar4 *pos = const_cast<uchar4 *>(tmp_pos);
        shift_color_pointer(&pos, buffer);
        colors[idx] = make_uchar4(pos->z, pos->y, pos->x, 0); // bgra

        // uchar4 rgbw = color_volume.ptr (VOLUME_Y * v.z + v.y)[v.x];
        // colors[idx] = make_uchar4 (rgbw.z, rgbw.y, rgbw.x, 0); //bgra
    }
}
} // namespace device
} // namespace pcl

void pcl::device::exctractColors(const PtrStep<uchar4> &color_volume,
                                 const pcl::gpu::tsdf_buffer *buffer,
                                 const float3 &volume_size,
                                 const PtrSz<PointType> &points,
                                 uchar4 *colors) {
    const int block = 256;
    float3 cell_size =
        make_float3(volume_size.x / VOLUME_X, volume_size.y / VOLUME_Y,
                    volume_size.z / VOLUME_Z);
    extractColorsKernel<<<divUp(points.size, block), block>>>(
        cell_size, color_volume, *buffer, points, colors);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
};

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void pcl::device::clearColorSlice(PtrStep<uchar4> color_volume,
                                  pcl::gpu::tsdf_buffer *buffer, int shiftX,
                                  int shiftY, int shiftZ) {
    int newX = buffer->origin_GRID.x + shiftX;
    int newY = buffer->origin_GRID.y + shiftY;

    int3 minBounds, maxBounds;

    // X
    if (newX >= 0) {
        minBounds.x = buffer->origin_GRID.x;
        maxBounds.x = newX;
    } else {
        minBounds.x = newX + buffer->voxels_size.x;
        maxBounds.x = buffer->origin_GRID.x + buffer->voxels_size.x;
    }

    if (minBounds.x > maxBounds.x)
        std::swap(minBounds.x, maxBounds.x);

    // Y
    if (newY >= 0) {
        minBounds.y = buffer->origin_GRID.y;
        maxBounds.y = newY;
    } else {
        minBounds.y = newY + buffer->voxels_size.y;
        maxBounds.y = buffer->origin_GRID.y + buffer->voxels_size.y;
    }

    if (minBounds.y > maxBounds.y)
        std::swap(minBounds.y, maxBounds.y);

    // Z
    minBounds.z = buffer->origin_GRID.z;
    maxBounds.z = shiftZ;

    // call kernel
    dim3 block(32, 16);
    dim3 grid(1, 1, 1);
    grid.x = divUp(buffer->voxels_size.x, block.x);
    grid.y = divUp(buffer->voxels_size.y, block.y);

    clearColorSliceKernel<<<grid, block>>>(color_volume, *buffer, minBounds,
                                           maxBounds);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
}
